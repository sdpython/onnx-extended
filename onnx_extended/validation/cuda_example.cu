#include "hip/hip_runtime.h"
#include "cuda_example.cuh"
#include "cuda_utils.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>

// https://github.com/mark-poscablo/gpu-sum-reduction/blob/master/sum_reduction/reduce.cu
// https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

namespace cuda_example {

template <unsigned int blockSize>
__device__ void warpReduce(volatile float *sdata, unsigned int tid) {
  if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
  if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
  if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
  if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
  if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
  if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void block_sum_reduce(float *g_idata, float *g_odata, unsigned int n) {
  extern __shared__ float sdata[];
  
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockSize*2) + tid;
  unsigned int gridSize = blockSize*2*gridDim.x;
  sdata[tid] = 0;

  while (i < n) {
    sdata[tid] += g_idata[i] + g_idata[i+blockSize];
    i += gridSize; 
  }
  __syncthreads();

  if (blockSize >= 512) { 
    if (tid < 256) { 
      sdata[tid] += sdata[tid + 256]; 
    }
    __syncthreads();
  }
  if (blockSize >= 256) { 
    if (tid < 128) { 
      sdata[tid] += sdata[tid + 128]; 
    }
    __syncthreads();
  }
  if (blockSize >= 128) { 
    if (tid < 64) { 
      sdata[tid] += sdata[tid + 64]; 
    } 
    __syncthreads(); 
  }
  if (tid < 32) {
    warpReduce<blockSize>(sdata, tid);
  }
  if (tid == 0) {
    g_odata[blockIdx.x] = sdata[0];
  }
}

float kernel_vector_sum_reduce(float* d_in, unsigned int d_in_len) {
	float total_sum = 0;

	constexpr unsigned int block_sz = 512; // maximum number of thread
	constexpr unsigned int max_elems_per_block = block_sz * 2;
	
	unsigned int grid_sz = 0;
	if (d_in_len <= max_elems_per_block) {
		grid_sz = (unsigned int)std::ceil(float(d_in_len) / float(max_elems_per_block));
	}
	else {
		grid_sz = d_in_len / max_elems_per_block;
		if (d_in_len % max_elems_per_block != 0)
			grid_sz++;
	}

	float* d_block_sums;
	checkCudaErrors(hipMalloc(&d_block_sums, sizeof(float) * grid_sz));
	checkCudaErrors(hipMemset(d_block_sums, 0, sizeof(float) * grid_sz));

	block_sum_reduce<max_elems_per_block><<<grid_sz, block_sz>>>(d_block_sums, d_in, d_in_len);

	if (grid_sz <= max_elems_per_block) {
		float* d_total_sum;
		checkCudaErrors(hipMalloc(&d_total_sum, sizeof(unsigned int)));
		checkCudaErrors(hipMemset(d_total_sum, 0, sizeof(unsigned int)));
		block_sum_reduce<max_elems_per_block><<<1, block_sz>>>(d_total_sum, d_block_sums, grid_sz);
		//reduce4<<<1, block_sz, sizeof(unsigned int) * block_sz>>>(d_total_sum, d_block_sums, grid_sz);
		checkCudaErrors(hipMemcpy(&total_sum, d_total_sum, sizeof(unsigned int), hipMemcpyDeviceToHost));
		checkCudaErrors(hipFree(d_total_sum));
	}
	else {
		float* d_in_block_sums;
		checkCudaErrors(hipMalloc(&d_in_block_sums, sizeof(unsigned int) * grid_sz));
		checkCudaErrors(hipMemcpy(d_in_block_sums, d_block_sums, sizeof(unsigned int) * grid_sz, hipMemcpyDeviceToDevice));
		total_sum = kernel_vector_sum_reduce(d_in_block_sums, grid_sz);
		checkCudaErrors(hipFree(d_in_block_sums));
	}

	checkCudaErrors(hipFree(d_block_sums));
	return total_sum;
}

} // namespace cuda_example
