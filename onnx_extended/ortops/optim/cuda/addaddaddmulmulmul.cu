#include "hip/hip_runtime.h"
#include "addaddaddmulmulmul.h"
#include "common/c_op_helpers.h"
#include "common/common_kernels.h"
#include "cuda/common_kernels_cuda.h"
#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>

namespace ortops {

#ifndef CUDA_LONG
#define CUDA_LONG int32_t
#endif

struct GridDim {
  enum : CUDA_LONG {
    maxThreadsPerBlock = 256, // max threads per block
    maxElementsPerThread = 4, // max element processed per thread
  };
};

__device__ __forceinline__ void _add4_op(float *address, const float a, const float b,
                                         const float c, const float d) {
  *address = a + b + c + d;
}

__device__ __forceinline__ void _add4_op(half *address, const half a, const half b,
                                         const half c, const half d) {
#if __CUDA_ARCH__ < 700
  *address =
      __float2half(__half2float(a) + __half2float(b) + __half2float(c) + __half2float(d));
#else
  *address = a + b + c + d;
#endif
}

__device__ __forceinline__ void _mul4_op(float *address, const float a, const float b,
                                         const float c, const float d) {
  *address = a * b * c * d;
}

__device__ __forceinline__ void _mul4_op(half *address, const half a, const half b,
                                         const half c, const half d) {
#if __CUDA_ARCH__ < 700
  *address =
      __float2half(__half2float(a) * __half2float(b) * __half2float(c) * __half2float(d));
#else
  *address = a * b * c * d;
#endif
}

template <typename T> struct Mul4Op {
  __device__ __inline__ void operator()(T *address, const T a, const T b, const T c,
                                        const T d) const {
    _mul4_op(address, a, b, c, d);
  }
};

template <typename T> struct Add4Op {
  __device__ __inline__ void operator()(T *address, const T a, const T b, const T c,
                                        const T d) const {
    _add4_op(address, a, b, c, d);
  }
};

template <typename T, typename TFunc, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void _BinaryElementWiseSimple(T *output_data, const T *pA, const T *pB, const T *pC,
                                         const T *pD, CUDA_LONG N, const TFunc func) {
  CUDA_LONG start = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;
  CUDA_LONG id = start;
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      func(output_data + id, pA[id], pB[id], pC[id], pD[id]);
      id += NumThreadsPerBlock;
    }
  }
}

template <class INT, class INT2> inline __host__ __device__ INT CeilDiv(INT a, INT2 b) {
  return (INT)(((size_t)a + (size_t)b - 1) / (size_t)b);
}

template <typename T, typename TFunc>
void BinaryElementWiseNoBroadcastImpl(hipStream_t stream, T *output_data, const T *pA,
                                      const T *pB, const T *pC, const T *pD, size_t count,
                                      const TFunc func) {
  if (count == 0) // special case where there's a dim value of 0 in the output shape
    return;

  const int num_elements_per_thread = GridDim::maxElementsPerThread;
  const int num_threads_per_block = GridDim::maxThreadsPerBlock;

  int blocksPerGrid =
      static_cast<int>(CeilDiv(count, num_threads_per_block * num_elements_per_thread));
  CUDA_LONG N = static_cast<CUDA_LONG>(count);

  _BinaryElementWiseSimple<T, TFunc, num_threads_per_block, num_elements_per_thread>
      <<<blocksPerGrid, num_threads_per_block, 0, stream>>>(output_data, pA, pB, pC, pD, N,
                                                            func);
}

//////////////////
// AddAddAddMulMulMulOp...
//////////////////

template <typename T, bool addition>
void *AddAddAddMulMulMulOp<T, addition>::CreateKernel(const OrtApi &api,
                                                      const OrtKernelInfo *info) const {
  return std::make_unique<AddAddAddMulMulMulKernel<T, addition>>(api, info).release();
}

template <typename T, bool addition>
const char *AddAddAddMulMulMulOp<T, addition>::GetName() const {
  return addition ? "AddAddAdd" : "MulMulMul";
}

template <typename T, bool addition>
const char *AddAddAddMulMulMulOp<T, addition>::GetExecutionProviderType() const {
  return "CUDAExecutionProvider";
}

template <typename T, bool addition>
size_t AddAddAddMulMulMulOp<T, addition>::GetInputTypeCount() const {
  return 4;
};

template <typename T, bool addition>
ONNXTensorElementDataType
AddAddAddMulMulMulOp<T, addition>::GetInputType(std::size_t /* index */) const {
  return CTypeToOnnxType<T>().onnx_type();
}

template <typename T, bool addition>
ONNXTensorElementDataType
AddAddAddMulMulMulOp<T, addition>::GetOutputType(std::size_t /* index */) const {
  return CTypeToOnnxType<T>().onnx_type();
}

template <typename T, bool addition>
OrtCustomOpInputOutputCharacteristic
AddAddAddMulMulMulOp<T, addition>::GetInputCharacteristic(std::size_t index) const {
  switch (index) {
  case 0:
  case 1:
  case 2:
  case 3:
    return OrtCustomOpInputOutputCharacteristic::INPUT_OUTPUT_REQUIRED;
  default:
    EXT_THROW("Output index=", (uint64_t)index, " is out of boundary.");
  }
}

template <typename T, bool addition>
size_t AddAddAddMulMulMulOp<T, addition>::GetOutputTypeCount() const {
  return 1;
}

template <typename T, bool addition>
OrtCustomOpInputOutputCharacteristic
AddAddAddMulMulMulOp<T, addition>::GetOutputCharacteristic(std::size_t index) const {
  switch (index) {
  case 0:
    return OrtCustomOpInputOutputCharacteristic::INPUT_OUTPUT_REQUIRED;
  default:
    EXT_THROW("Output index=", (uint64_t)index, " is out of boundary.");
  }
}

///////////////////
// AddAddAddMulMulMulKernel
///////////////////

template <typename T, bool addition>
AddAddAddMulMulMulKernel<T, addition>::AddAddAddMulMulMulKernel(const OrtApi &api,
                                                                const OrtKernelInfo *info) {}

template <typename T, bool addition>
void AddAddAddMulMulMulKernel<T, addition>::Compute(OrtKernelContext *context) {
  Ort::KernelContext ctx(context);

  int n_inputs = ctx.GetInputCount();
  EXT_ENFORCE(n_inputs == 4, "Expected 3 inputs not ", n_inputs, ".");
  Ort::ConstValue A = ctx.GetInput(0);
  Ort::ConstValue B = ctx.GetInput(1);
  Ort::ConstValue C = ctx.GetInput(2);
  Ort::ConstValue D = ctx.GetInput(3);
  Ort::UnownedValue output;

  std::vector<int64_t> dimsA = A.GetTensorTypeAndShapeInfo().GetShape();
  std::vector<int64_t> dimsB = B.GetTensorTypeAndShapeInfo().GetShape();
  std::vector<int64_t> dimsC = C.GetTensorTypeAndShapeInfo().GetShape();
  std::vector<int64_t> dimsD = D.GetTensorTypeAndShapeInfo().GetShape();

  auto sizeA = onnx_c_ops::flattened_dimension(dimsA);
  auto sizeB = onnx_c_ops::flattened_dimension(dimsB);
  auto sizeC = onnx_c_ops::flattened_dimension(dimsC);
  auto sizeD = onnx_c_ops::flattened_dimension(dimsD);

  EXT_ENFORCE(sizeA == sizeB && sizeB == sizeC && sizeC == sizeD,
              "The kernel does not support broadcast.");

  hipStream_t hip_stream = (hipStream_t)ctx.GetGPUComputeStream();
  // CUDA_THROW_IF_ERROR(hipStreamSynchronize(hip_stream));

  output = ctx.GetOutput(0, dimsA);

  if (addition) {
    BinaryElementWiseNoBroadcastImpl(
        hip_stream, output.GetTensorMutableData<T>(), A.GetTensorData<T>(),
        B.GetTensorData<T>(), C.GetTensorData<T>(), D.GetTensorData<T>(), sizeA, Add4Op<T>());
  } else {
    BinaryElementWiseNoBroadcastImpl(
        hip_stream, output.GetTensorMutableData<T>(), A.GetTensorData<T>(),
        B.GetTensorData<T>(), C.GetTensorData<T>(), D.GetTensorData<T>(), sizeA, Mul4Op<T>());
  }
}

static AddAddAddMulMulMulOp<float, true> _add432;
static AddAddAddMulMulMulOp<half, true> _add416;
static AddAddAddMulMulMulOp<float, false> _mul432;
static AddAddAddMulMulMulOp<half, false> _mul416;

} // namespace ortops
