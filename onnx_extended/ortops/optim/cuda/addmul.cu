#include "hip/hip_runtime.h"
#include "addmul.h"
#include "common/c_op_helpers.h"
#include "common/common_kernels.h"
#include "cuda/common_kernels_cuda.h"
#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>

namespace ortops {

#ifndef CUDA_LONG
#define CUDA_LONG int32_t
#endif

struct GridDim {
  enum : CUDA_LONG {
    maxThreadsPerBlock = 256, // max threads per block
    maxElementsPerThread = 4, // max element processed per thread
  };
};

__device__ __forceinline__ void _addmul_op(float *address, const float a, const float b,
                                           const float c) {
  *address = (a + b) * c;
}

__device__ __forceinline__ void _addmul_op(half *address, const half a, const half b,
                                           const half c) {
#if __CUDA_ARCH__ < 700
  *address = __float2half((__half2float(a) + __half2float(b)) * __half2float(c));
#else
  *address = (a + b) * c;
#endif
}

__device__ __forceinline__ void _muladd_op(float *address, const float a, const float b,
                                           const float c) {
  *address = a * b + c;
}

__device__ __forceinline__ void _muladd_op(half *address, const half a, const half b,
                                           const half c) {
#if __CUDA_ARCH__ < 700
  *address = __float2half(__half2float(a) * __half2float(b) + __half2float(c));
#else
  *address = a * b + c;
#endif
}

template <typename T> struct AddMul {
  __device__ __inline__ void operator()(T *address, const T a, const T b, const T c) const {
    _addmul_op(address, a, b, c);
  }
};

template <typename T> struct MulAdd {
  __device__ __inline__ void operator()(T *address, const T a, const T b, const T c) const {
    _muladd_op(address, a, b, c);
  }
};

template <typename T, typename TFunc, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void _BinaryElementWiseSimple(T *output_data, const T *pA, const T *pB, const T *pC,
                                         CUDA_LONG nA, CUDA_LONG nB, CUDA_LONG nC, CUDA_LONG N,
                                         const TFunc func) {
  CUDA_LONG start = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;
  CUDA_LONG id = start;
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      func(output_data + id, pA[id % nA], pB[id % nB], pC[id % nC]);
      id += NumThreadsPerBlock;
    }
  }
}

template <typename T, typename TFunc, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void _BinaryElementWiseSimpleSwitchMiddle(T *output_data, const T *pA, const T *pB,
                                                     const T *pC, CUDA_LONG nA, CUDA_LONG nB,
                                                     CUDA_LONG nC, CUDA_LONG N,
                                                     const TFunc func, CUDA_LONG d2,
                                                     CUDA_LONG d3, CUDA_LONG d4) {
  CUDA_LONG start = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;
  CUDA_LONG id = start;
  CUDA_LONG k, j, ido;
  // dimension, d1, d2, d3, d4
  // indices i, j, k, l
  // [i,j,k,l] --> i d2*d3*d4 + j d3*d4 + k d4 + l
  // l = id % d4
  // k = (id // d4) % d3
  // j = (id // (d3*d4) % d2
  // [i,k,j,l] -> i d2*d3*d4 + k d2*d4 + j d4 + l
  //           -> i d2*d3*d4 + [(id // d4) % d3] d2*d4 + [(id // (d3*d4) % d2] d4 + l
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      k = (id / d4) % d3;
      j = (id / (d4 * d3)) % d2;
      ido = id + d4 * ((k * d2 + j) - (j * d3 + k));
      func(output_data + ido, pA[id % nA], pB[id % nB], pC[id % nC]);
      id += NumThreadsPerBlock;
    }
  }
}

template <class INT, class INT2> inline __host__ __device__ INT CeilDiv(INT a, INT2 b) {
  return (INT)(((size_t)a + (size_t)b - 1) / (size_t)b);
}

template <typename T, typename TFunc>
void BinaryElementWiseImpl(hipStream_t stream, T *output_data, const T *pA, const T *pB,
                           const T *pC, int64_t countA, int64_t countB, int64_t countC,
                           int64_t max_count, const TFunc func) {
  if (max_count == 0) // special case where there's a dim value of 0 in the output shape
    return;

  const int num_elements_per_thread = GridDim::maxElementsPerThread;
  const int num_threads_per_block = GridDim::maxThreadsPerBlock;

  int blocksPerGrid =
      static_cast<int>(CeilDiv(max_count, num_threads_per_block * num_elements_per_thread));

  _BinaryElementWiseSimple<T, TFunc, num_threads_per_block, num_elements_per_thread>
      <<<blocksPerGrid, num_threads_per_block, 0, stream>>>(
          output_data, pA, pB, pC, static_cast<CUDA_LONG>(countA),
          static_cast<CUDA_LONG>(countB), static_cast<CUDA_LONG>(countC),
          static_cast<CUDA_LONG>(max_count), func);
}

template <typename T, typename TFunc>
void BinaryElementWiseImplSwitchMiddle(hipStream_t stream, T *output_data, const T *pA,
                                       const T *pB, const T *pC, int64_t countA, int64_t countB,
                                       int64_t countC, int64_t max_count, const TFunc func,
                                       int64_t d2, int64_t d3, int64_t d4) {
  if (max_count == 0) // special case where there's a dim value of 0 in the output shape
    return;

  const int num_elements_per_thread = GridDim::maxElementsPerThread;
  const int num_threads_per_block = GridDim::maxThreadsPerBlock;

  int blocksPerGrid =
      static_cast<int>(CeilDiv(max_count, num_threads_per_block * num_elements_per_thread));

  _BinaryElementWiseSimpleSwitchMiddle<T, TFunc, num_threads_per_block, num_elements_per_thread>
      <<<blocksPerGrid, num_threads_per_block, 0, stream>>>(
          output_data, pA, pB, pC, static_cast<CUDA_LONG>(countA),
          static_cast<CUDA_LONG>(countB), static_cast<CUDA_LONG>(countC),
          static_cast<CUDA_LONG>(max_count), func, static_cast<CUDA_LONG>(d2),
          static_cast<CUDA_LONG>(d3), static_cast<CUDA_LONG>(d4));
}

//////////////////
// AddMulOp...
//////////////////

template <typename T, bool addition>
void *AddMulOp<T, addition>::CreateKernel(const OrtApi &api, const OrtKernelInfo *info) const {
  return std::make_unique<AddMulKernel<T, addition>>(api, info).release();
}

template <typename T, bool addition> const char *AddMulOp<T, addition>::GetName() const {
  return addition ? "AddMul" : "MulAdd";
}

template <typename T, bool addition>
const char *AddMulOp<T, addition>::GetExecutionProviderType() const {
  return "CUDAExecutionProvider";
}

template <typename T, bool addition> size_t AddMulOp<T, addition>::GetInputTypeCount() const {
  return 3;
};

template <typename T, bool addition>
ONNXTensorElementDataType AddMulOp<T, addition>::GetInputType(std::size_t /* index */) const {
  return CTypeToOnnxType<T>().onnx_type();
}

template <typename T, bool addition>
ONNXTensorElementDataType AddMulOp<T, addition>::GetOutputType(std::size_t /* index */) const {
  return CTypeToOnnxType<T>().onnx_type();
}

template <typename T, bool addition>
OrtCustomOpInputOutputCharacteristic
AddMulOp<T, addition>::GetInputCharacteristic(std::size_t index) const {
  switch (index) {
  case 0:
  case 1:
  case 2:
    return OrtCustomOpInputOutputCharacteristic::INPUT_OUTPUT_REQUIRED;
  default:
    EXT_THROW("Output index=", (uint64_t)index, " is out of boundary.");
  }
}

template <typename T, bool addition> size_t AddMulOp<T, addition>::GetOutputTypeCount() const {
  return 1;
}

template <typename T, bool addition>
OrtCustomOpInputOutputCharacteristic
AddMulOp<T, addition>::GetOutputCharacteristic(std::size_t index) const {
  switch (index) {
  case 0:
    return OrtCustomOpInputOutputCharacteristic::INPUT_OUTPUT_REQUIRED;
  default:
    EXT_THROW("Output index=", (uint64_t)index, " is out of boundary.");
  }
}

///////////////////
// AddMulKernel
///////////////////

template <typename T, bool addition>
AddMulKernel<T, addition>::AddMulKernel(const OrtApi &api, const OrtKernelInfo *info) {
  switch_middle_axis_ =
      KernelInfoGetOptionalAttributeInt64AsBool(api, info, "transposeMiddle", false);
}

template <typename T, bool addition>
void AddMulKernel<T, addition>::Compute(OrtKernelContext *context) {
  Ort::KernelContext ctx(context);

  int n_inputs = ctx.GetInputCount();
  EXT_ENFORCE(n_inputs == 3, "Expected 3 inputs not ", n_inputs, ".");
  Ort::ConstValue A = ctx.GetInput(0);
  Ort::ConstValue B = ctx.GetInput(1);
  Ort::ConstValue C = ctx.GetInput(2);
  Ort::UnownedValue output;

  std::vector<int64_t> dimsA = A.GetTensorTypeAndShapeInfo().GetShape();
  std::vector<int64_t> dimsB = B.GetTensorTypeAndShapeInfo().GetShape();
  std::vector<int64_t> dimsC = C.GetTensorTypeAndShapeInfo().GetShape();

  auto sizeA = onnx_c_ops::flattened_dimension(dimsA);
  auto sizeB = onnx_c_ops::flattened_dimension(dimsB);
  auto sizeC = onnx_c_ops::flattened_dimension(dimsC);
  auto max_size = std::max(sizeA, std::max(sizeB, sizeC));

  auto max_rank = std::max(dimsA.size(), std::max(dimsB.size(), dimsC.size()));
  while (dimsA.size() < max_rank)
    dimsA.insert(dimsA.begin(), 1);
  while (dimsB.size() < max_rank)
    dimsB.insert(dimsB.begin(), 1);
  while (dimsC.size() < max_rank)
    dimsC.insert(dimsC.begin(), 1);

  hipStream_t hip_stream = (hipStream_t)ctx.GetGPUComputeStream();
  // CUDA_THROW_IF_ERROR(hipStreamSynchronize(hip_stream));

  std::vector<int64_t> output_dims(dimsA.size());
  for (size_t i = 0; i < dimsA.size(); ++i) {
    output_dims[i] = std::max(std::max(dimsA[i], dimsB[i]), dimsC[i]);
  }

  if (switch_middle_axis_) {
    EXT_ENFORCE(output_dims.size() == 4,
                "transposeMiddle is true but the output does not have 4 dimensions but ",
                output_dims.size(), ".");
    int64_t d4 = output_dims[output_dims.size() - 1];
    int64_t d3 = output_dims[output_dims.size() - 2];
    int64_t d2 = output_dims[output_dims.size() - 3];
    output_dims[1] = d3;
    output_dims[2] = d2;
    output = ctx.GetOutput(0, output_dims);
    if (addition) {
      BinaryElementWiseImplSwitchMiddle(hip_stream, output.GetTensorMutableData<T>(),
                                        A.GetTensorData<T>(), B.GetTensorData<T>(),
                                        C.GetTensorData<T>(), sizeA, sizeB, sizeC, max_size,
                                        AddMul<T>(), d2, d3, d4);
    } else {
      BinaryElementWiseImplSwitchMiddle(hip_stream, output.GetTensorMutableData<T>(),
                                        A.GetTensorData<T>(), B.GetTensorData<T>(),
                                        C.GetTensorData<T>(), sizeA, sizeB, sizeC, max_size,
                                        MulAdd<T>(), d2, d3, d4);
    }
  } else if (addition) {
    output = ctx.GetOutput(0, output_dims);
    BinaryElementWiseImpl(hip_stream, output.GetTensorMutableData<T>(), A.GetTensorData<T>(),
                          B.GetTensorData<T>(), C.GetTensorData<T>(), sizeA, sizeB, sizeC,
                          max_size, AddMul<T>());
  } else {
    output = ctx.GetOutput(0, output_dims);
    BinaryElementWiseImpl(hip_stream, output.GetTensorMutableData<T>(), A.GetTensorData<T>(),
                          B.GetTensorData<T>(), C.GetTensorData<T>(), sizeA, sizeB, sizeC,
                          max_size, MulAdd<T>());
  }
}

static AddMulOp<float, true> _addmul32;
static AddMulOp<half, true> _addmul16;
static AddMulOp<float, false> _muladd32;
static AddMulOp<half, false> _muladd16;

} // namespace ortops
