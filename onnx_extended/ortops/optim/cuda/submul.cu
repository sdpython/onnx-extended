#include "hip/hip_runtime.h"
#include "common/c_op_helpers.h"
#include "common/common_kernels.h"
#include "cuda/common_kernels_cuda.h"
#include "submul.h"
#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>

namespace ortops {

#ifndef CUDA_LONG
#define CUDA_LONG int32_t
#endif

struct GridDim {
  enum : CUDA_LONG {
    maxThreadsPerBlock = 256, // max threads per block
    maxElementsPerThread = 4, // max element processed per thread
  };
};

__device__ __forceinline__ void _submul_op(float *address, const float a, const float b,
                                           const float c) {
  *address = (a - b) * c;
}

__device__ __forceinline__ void _submul_op(half *address, const half a, const half b,
                                           const half c) {
#if __CUDA_ARCH__ < 700
  *address = __float2half((__half2float(a) - __half2float(b)) * __half2float(c));
#else
  *address = (a - b) * c;
#endif
}

__device__ __forceinline__ void _submul_neg_op(float *address, const float a, const float b,
                                               const float c) {
  *address = (b - a) * c;
}

__device__ __forceinline__ void _submul_neg_op(half *address, const half a, const half b,
                                               const half c) {
#if __CUDA_ARCH__ < 700
  *address = __float2half((__half2float(b) - __half2float(a)) * __half2float(c));
#else
  *address = (b - a) * c;
#endif
}

__device__ __forceinline__ void _mulsub_op(float *address, const float a, const float b,
                                           const float c) {
  *address = a * b - c;
}

__device__ __forceinline__ void _mulsub_op(half *address, const half a, const half b,
                                           const half c) {
#if __CUDA_ARCH__ < 700
  *address = __float2half(__half2float(a) * __half2float(b) - __half2float(c));
#else
  *address = a * b - c;
#endif
}

__device__ __forceinline__ void _mulsub_neg_op(float *address, const float a, const float b,
                                               const float c) {
  *address = c - a * b;
}

__device__ __forceinline__ void _mulsub_neg_op(half *address, const half a, const half b,
                                               const half c) {
#if __CUDA_ARCH__ < 700
  *address = __float2half(__half2float(c) - __half2float(a) * __half2float(b));
#else
  *address = c - a * b;
#endif
}

template <typename T> struct SubMul {
  __device__ __inline__ void operator()(T *address, const T a, const T b, const T c) const {
    _submul_op(address, a, b, c);
  }
};

template <typename T> struct MulSub {
  __device__ __inline__ void operator()(T *address, const T a, const T b, const T c) const {
    _mulsub_op(address, a, b, c);
  }
};

template <typename T> struct SubMulNeg {
  __device__ __inline__ void operator()(T *address, const T a, const T b, const T c) const {
    _submul_neg_op(address, a, b, c);
  }
};

template <typename T> struct MulSubNeg {
  __device__ __inline__ void operator()(T *address, const T a, const T b, const T c) const {
    _mulsub_neg_op(address, a, b, c);
  }
};

template <typename T, typename TFunc, int NumThreadsPerBlock, int NumElementsPerThread>
__global__ void _BinaryElementWiseSimple(T *output_data, const T *pA, const T *pB, const T *pC,
                                         CUDA_LONG nA, CUDA_LONG nB, CUDA_LONG nC, CUDA_LONG N,
                                         const TFunc func) {
  CUDA_LONG start = NumElementsPerThread * NumThreadsPerBlock * blockIdx.x + threadIdx.x;
  CUDA_LONG id = start;
#pragma unroll
  for (int i = 0; i < NumElementsPerThread; i++) {
    if (id < N) {
      func(output_data + id, pA[id % nA], pB[id % nB], pC[id % nC]);
      id += NumThreadsPerBlock;
    }
  }
}

template <class INT, class INT2> inline __host__ __device__ INT CeilDiv(INT a, INT2 b) {
  return (INT)(((size_t)a + (size_t)b - 1) / (size_t)b);
}

template <typename T, typename TFunc>
void BinaryElementWiseNoBroadcastImpl(hipStream_t stream, T *output_data, const T *pA,
                                      const T *pB, const T *pC, int64_t countA, int64_t countB,
                                      int64_t countC, int64_t max_count, const TFunc func) {
  if (max_count == 0) // special case where there's a dim value of 0 in the output shape
    return;

  const int num_elements_per_thread = GridDim::maxElementsPerThread;
  const int num_threads_per_block = GridDim::maxThreadsPerBlock;

  int blocksPerGrid =
      static_cast<int>(CeilDiv(max_count, num_threads_per_block * num_elements_per_thread));

  _BinaryElementWiseSimple<T, TFunc, num_threads_per_block, num_elements_per_thread>
      <<<blocksPerGrid, num_threads_per_block, 0, stream>>>(
          output_data, pA, pB, pC, static_cast<CUDA_LONG>(countA),
          static_cast<CUDA_LONG>(countB), static_cast<CUDA_LONG>(countC),
          static_cast<CUDA_LONG>(max_count), func);
}

//////////////////
// SubMulOp...
//////////////////

template <typename T, bool addition>
void *SubMulOp<T, addition>::CreateKernel(const OrtApi &api, const OrtKernelInfo *info) const {
  return std::make_unique<SubMulKernel<T, addition>>(api, info).release();
}

template <typename T, bool addition> const char *SubMulOp<T, addition>::GetName() const {
  return addition ? "SubMul" : "MulSub";
}

template <typename T, bool addition>
const char *SubMulOp<T, addition>::GetExecutionProviderType() const {
  return "CUDAExecutionProvider";
}

template <typename T, bool addition> size_t SubMulOp<T, addition>::GetInputTypeCount() const {
  return 3;
};

template <typename T, bool addition>
ONNXTensorElementDataType SubMulOp<T, addition>::GetInputType(std::size_t /* index */) const {
  return CTypeToOnnxType<T>().onnx_type();
}

template <typename T, bool addition>
ONNXTensorElementDataType SubMulOp<T, addition>::GetOutputType(std::size_t /* index */) const {
  return CTypeToOnnxType<T>().onnx_type();
}

template <typename T, bool addition>
OrtCustomOpInputOutputCharacteristic
SubMulOp<T, addition>::GetInputCharacteristic(std::size_t index) const {
  switch (index) {
  case 0:
  case 1:
  case 2:
    return OrtCustomOpInputOutputCharacteristic::INPUT_OUTPUT_REQUIRED;
  default:
    EXT_THROW("Output index=", (uint64_t)index, " is out of boundary.");
  }
}

template <typename T, bool addition> size_t SubMulOp<T, addition>::GetOutputTypeCount() const {
  return 1;
}

template <typename T, bool addition>
OrtCustomOpInputOutputCharacteristic
SubMulOp<T, addition>::GetOutputCharacteristic(std::size_t index) const {
  switch (index) {
  case 0:
    return OrtCustomOpInputOutputCharacteristic::INPUT_OUTPUT_REQUIRED;
  default:
    EXT_THROW("Output index=", (uint64_t)index, " is out of boundary.");
  }
}

///////////////////
// SubMulKernel
///////////////////

template <typename T, bool addition>
SubMulKernel<T, addition>::SubMulKernel(const OrtApi &api, const OrtKernelInfo *info) {
  negative_ = KernelInfoGetOptionalAttributeInt64AsBool(api, info, "negative", false);
}

template <typename T, bool addition>
void SubMulKernel<T, addition>::Compute(OrtKernelContext *context) {
  Ort::KernelContext ctx(context);

  int n_inputs = ctx.GetInputCount();
  EXT_ENFORCE(n_inputs == 3, "Expected 3 inputs not ", n_inputs, ".");
  Ort::ConstValue A = ctx.GetInput(0);
  Ort::ConstValue B = ctx.GetInput(1);
  Ort::ConstValue C = ctx.GetInput(2);
  Ort::UnownedValue output;

  std::vector<int64_t> dimsA = A.GetTensorTypeAndShapeInfo().GetShape();
  std::vector<int64_t> dimsB = B.GetTensorTypeAndShapeInfo().GetShape();
  std::vector<int64_t> dimsC = C.GetTensorTypeAndShapeInfo().GetShape();

  auto sizeA = onnx_c_ops::flattened_dimension(dimsA);
  auto sizeB = onnx_c_ops::flattened_dimension(dimsB);
  auto sizeC = onnx_c_ops::flattened_dimension(dimsC);
  auto max_size = std::max(sizeA, std::max(sizeB, sizeC));

  auto max_rank = std::max(dimsA.size(), std::max(dimsB.size(), dimsC.size()));
  while (dimsA.size() < max_rank)
    dimsA.insert(dimsA.begin(), 1);
  while (dimsB.size() < max_rank)
    dimsB.insert(dimsB.begin(), 1);
  while (dimsC.size() < max_rank)
    dimsC.insert(dimsC.begin(), 1);

  hipStream_t hip_stream = (hipStream_t)ctx.GetGPUComputeStream();
  // CUDA_THROW_IF_ERROR(hipStreamSynchronize(hip_stream));

  std::vector<int64_t> output_dims(dimsA.size());
  for (size_t i = 0; i < dimsA.size(); ++i) {
    output_dims[i] = std::max(std::max(dimsA[i], dimsB[i]), dimsC[i]);
  }
  output = ctx.GetOutput(0, output_dims);

  if (addition) {
    if (negative_) {
      BinaryElementWiseNoBroadcastImpl(hip_stream, output.GetTensorMutableData<T>(),
                                       A.GetTensorData<T>(), B.GetTensorData<T>(),
                                       C.GetTensorData<T>(), sizeA, sizeB, sizeC, max_size,
                                       SubMulNeg<T>());
    } else {
      BinaryElementWiseNoBroadcastImpl(hip_stream, output.GetTensorMutableData<T>(),
                                       A.GetTensorData<T>(), B.GetTensorData<T>(),
                                       C.GetTensorData<T>(), sizeA, sizeB, sizeC, max_size,
                                       SubMul<T>());
    }
  } else {
    if (negative_) {
      BinaryElementWiseNoBroadcastImpl(hip_stream, output.GetTensorMutableData<T>(),
                                       A.GetTensorData<T>(), B.GetTensorData<T>(),
                                       C.GetTensorData<T>(), sizeA, sizeB, sizeC, max_size,
                                       MulSubNeg<T>());
    } else {
      BinaryElementWiseNoBroadcastImpl(hip_stream, output.GetTensorMutableData<T>(),
                                       A.GetTensorData<T>(), B.GetTensorData<T>(),
                                       C.GetTensorData<T>(), sizeA, sizeB, sizeC, max_size,
                                       MulSub<T>());
    }
  }
}

static SubMulOp<float, true> _submul32;
static SubMulOp<half, true> _submul16;
static SubMulOp<float, false> _mulsub32;
static SubMulOp<half, false> _mulsub16;

} // namespace ortops
