#include "hip/hip_runtime.h"
#include "common/common_kernels_cuda.h"
#include "custom_gemm.h"
#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#if ORT_VERSION >= 1160 && CUDA_VERSION >= 11080
#include <hip/hip_fp8.h>
#endif

// see
// https://gitlab.com/nvidia/headers/cuda-individual/cublas/-/blob/main/hipblaslt.h

namespace ortops {

////////////////////
// CustomGemmOpFloat
////////////////////

void *CustomGemmOpFloat::CreateKernel(const OrtApi &api,
                                      const OrtKernelInfo *info) const {
  return std::make_unique<CustomGemmKernel>(api, info).release();
};

const char *CustomGemmOpFloat::GetName() const { return "CustomGemmFloat"; };

const char *CustomGemmOpFloat::GetExecutionProviderType() const {
  return "CUDAExecutionProvider";
};

size_t CustomGemmOpFloat::GetInputTypeCount() const { return 2; };

ONNXTensorElementDataType CustomGemmOpFloat::GetInputType(size_t index) const {
  return ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT;
};

size_t CustomGemmOpFloat::GetOutputTypeCount() const { return 1; };

ONNXTensorElementDataType CustomGemmOpFloat::GetOutputType(size_t index) const {
  return ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT;
};

///////////////////////////
// CustomGemmOpFloat8E4M3FN
///////////////////////////

#if ORT_VERSION >= 1160 && CUDA_VERSION >= 11080

void *CustomGemmOpFloat8E4M3FN::CreateKernel(const OrtApi &api,
                                             const OrtKernelInfo *info) const {
  return std::make_unique<CustomGemmKernel>(api, info).release();
};

const char *CustomGemmOpFloat8E4M3FN::GetName() const {
  return "CustomGemmFloat8E4M3FN";
};

const char *CustomGemmOpFloat8E4M3FN::GetExecutionProviderType() const {
  return "CUDAExecutionProvider";
};

size_t CustomGemmOpFloat8E4M3FN::GetInputTypeCount() const { return 4; };

ONNXTensorElementDataType
CustomGemmOpFloat8E4M3FN::GetInputType(size_t index) const {
  switch (index) {
  case 0: // A
  case 1: // B
    return ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT8E4M3FN;
  case 2: // scale A
  case 3: // scale B
    return ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT;
  default:
    EXT_THROW("index=", index, " is out of boundary.");
  }
};

size_t CustomGemmOpFloat8E4M3FN::GetOutputTypeCount() const { return 2; };

ONNXTensorElementDataType
CustomGemmOpFloat8E4M3FN::GetOutputType(size_t index) const {
  // D, scale D
  switch (index) {
  case 0:
    return ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT16;
  case 1:
    return ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT;
  default:
    EXT_THROW("index=", index, " is out of boundary.");
  }
}

#endif

///////////////////
// CustomGemmKernel
///////////////////

CustomGemmKernel::CustomGemmKernel(const OrtApi &api,
                                   const OrtKernelInfo *info) {
  ThrowOnError(api, api.KernelInfoGetAttribute_float(info, "alpha", &alpha_));
  // ThrowOnError(api, api.KernelInfoGetAttribute_float(info, "beta", &beta_));
  row_major_ =
      KernelInfoGetOptionalAttributeInt64AsBool(api, info, "row_major", true);
  transA_ =
      KernelInfoGetOptionalAttributeInt64AsBool(api, info, "transA", false);
  transB_ =
      KernelInfoGetOptionalAttributeInt64AsBool(api, info, "transB", false);
  fastAccumulationMode_ = KernelInfoGetOptionalAttributeInt64AsBool(
      api, info, "fastAccumulationMode", false);
  smCount_ = KernelInfoGetOptionalAttributeInt64(api, info, "smCount", 0);

  // A string attribute.
  std::string compute_type = KernelInfoGetOptionalAttributeString(
      api, info, "computeType", "HIPBLAS_COMPUTE_32F");
  if (compute_type == "HIPBLAS_COMPUTE_16F") {
    computeType_ = HIPBLAS_COMPUTE_16F;
  } else if (compute_type == "HIPBLAS_COMPUTE_32F") {
    computeType_ = HIPBLAS_COMPUTE_32F;
  } else if (compute_type == "HIPBLAS_COMPUTE_32F_FAST_16F") {
    computeType_ = HIPBLAS_COMPUTE_32F_FAST_16F;
  } else if (compute_type == "HIPBLAS_COMPUTE_32F_FAST_16BF") {
    computeType_ = HIPBLAS_COMPUTE_32F_FAST_16BF;
  } else if (compute_type == "HIPBLAS_COMPUTE_32F_FAST_TF32") {
    computeType_ = HIPBLAS_COMPUTE_32F_FAST_TF32;
  } else {
    EXT_THROW("Unexpected value for compute_type '", compute_type, "'.");
  }
}

void CustomGemmKernel::set(int M, int N, int K, int &lda, int &ldb,
                           int &ldd) const {
  if (transA_ && !transB_) { // TN
    lda = K;
    ldb = K;
    ldd = M;
  } else if (!transA_ && !transB_) { // NN
    lda = M;
    ldb = K;
    ldd = M;
  } else if (!transA_ && transB_) { // NT
    lda = M;
    ldb = N;
    ldd = M;
  } else { // TT
    EXT_THROW("transA_ == true && transB_ == true not allowed.");
  }
}

void CustomGemmKernel::Compute(OrtKernelContext *context) {
  Ort::KernelContext ctx(context);
  Ort::ConstValue input_A = ctx.GetInput(0);
  Ort::ConstValue input_B = ctx.GetInput(1);
  Ort::ConstValue scale_A, scale_B;

  auto memA = input_A.GetTensorMemoryInfo();
  EXT_ENFORCE(memA.GetDeviceType() ==
                  OrtMemoryInfoDeviceType::OrtMemoryInfoDeviceType_GPU,
              "Input A is not on CUDA");
  auto memB = input_B.GetTensorMemoryInfo();
  EXT_ENFORCE(memB.GetDeviceType() ==
                  OrtMemoryInfoDeviceType::OrtMemoryInfoDeviceType_GPU,
              "Input B is not on CUDA");

  int n_inputs = ctx.GetInputCount();
  if (n_inputs == 4) {
    scale_A = ctx.GetInput(2);
    scale_B = ctx.GetInput(3);
    auto memsA = scale_A.GetTensorMemoryInfo();
    EXT_ENFORCE(memsA.GetDeviceType() ==
                    OrtMemoryInfoDeviceType::OrtMemoryInfoDeviceType_GPU,
                "Scale A is not on CUDA");
    auto memsB = scale_B.GetTensorMemoryInfo();
    EXT_ENFORCE(memsB.GetDeviceType() ==
                    OrtMemoryInfoDeviceType::OrtMemoryInfoDeviceType_GPU,
                "Scale B is not on CUDA");
  } else if (n_inputs != 2) {
    EXT_THROW("Number of inputs must be 2 or 4.");
  }

  std::vector<int64_t> a_shape = input_A.GetTensorTypeAndShapeInfo().GetShape();
  std::vector<int64_t> b_shape = input_B.GetTensorTypeAndShapeInfo().GetShape();

  EXT_ENFORCE(a_shape.size() == 2);
  EXT_ENFORCE(b_shape.size() == 2);

  ONNXTensorElementDataType dtypes[4] = {
      input_A.GetTensorTypeAndShapeInfo().GetElementType(),
      input_B.GetTensorTypeAndShapeInfo().GetElementType(),
      n_inputs == 4 ? scale_A.GetTensorTypeAndShapeInfo().GetElementType()
                    : ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT,
      n_inputs == 4 ? scale_B.GetTensorTypeAndShapeInfo().GetElementType()
                    : ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT,
  };

  int M, N, K;
  if (transA_) {
    M = a_shape[1];
    K = a_shape[0];
  } else {
    M = a_shape[0];
    K = a_shape[1];
  }

  N = transB_ ? b_shape[0] : b_shape[1];
  EXT_ENFORCE(M >= 0 && K > 0 && N >= 0);

  std::vector<int64_t> dimensions{M, N};
  Ort::UnownedValue Y = ctx.GetOutput(0, dimensions);
  ONNXTensorElementDataType out_dtype =
      Y.GetTensorTypeAndShapeInfo().GetElementType();
  auto memY = Y.GetTensorMemoryInfo();
  EXT_ENFORCE(memY.GetDeviceType() ==
                  OrtMemoryInfoDeviceType::OrtMemoryInfoDeviceType_GPU,
              "Output 1 is not on CUDA");

  hipStream_t stream = (hipStream_t)ctx.GetGPUComputeStream();
  hipblasLtHandle_t cublasLt;
  CUBLAS_THROW_IF_ERROR(hipblasLtCreate(&cublasLt));

  // #if defined(CUDA_VERSION) && CUDA_VERSION >= 11080
  int lda, ldb, ldd;
  set(M, N, K, lda, ldb, ldd);

  // Gemm, note that CUDA assumes col-major.
  // so Y(N,M) = alpha * op(B) x op(A).
  hipblasLtMatmulDesc_t operationDesc = nullptr;
  hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr,
                         Ddesc = nullptr;

  // Create matrix descriptors. Not setting any extra attributes.
  hipDataType a_cuda_type = ToCudaDataType(dtypes[0]);
  hipDataType b_cuda_type = ToCudaDataType(dtypes[1]);
  hipDataType d_cuda_type = ToCudaDataType(out_dtype);
  hipDataType bias_cuda_type =
      ToCudaDataType(ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT);
  hipDataType scale_cuda_type = bias_cuda_type;

  CUBLAS_THROW_IF_ERROR(hipblasLtMatrixLayoutCreate(
      &Adesc, a_cuda_type, transA_ ? M : K, transA_ ? K : M, lda));
  CUBLAS_THROW_IF_ERROR(hipblasLtMatrixLayoutCreate(
      &Bdesc, b_cuda_type, transB_ ? K : N, transB_ ? N : K, ldb));
  CUBLAS_THROW_IF_ERROR(
      hipblasLtMatrixLayoutCreate(&Ddesc, d_cuda_type, M, N, ldd));

  CUBLAS_THROW_IF_ERROR(
      hipblasLtMatmulDescCreate(&operationDesc, computeType_, scale_cuda_type));
  hipblasOperation_t transa = transA_ ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t transb = transB_ ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_THROW_IF_ERROR(hipblasLtMatmulDescSetAttribute(
      operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
  CUBLAS_THROW_IF_ERROR(hipblasLtMatmulDescSetAttribute(
      operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

  if (smCount_ != 0) {
    int math_sm_count = static_cast<int>(smCount_);
    CUBLAS_THROW_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, CUBLASLT_MATMUL_DESC_SM_COUNT_TARGET, &math_sm_count,
        sizeof(math_sm_count)));
  }

  const void *p_scale_a = nullptr;
  const void *p_scale_b = nullptr;
  const void *p_scale_d = nullptr;
  const void *p_scale_before = nullptr;
  if (n_inputs == 4) {
    // gemm float 8
    const int8_t ifast_accumulation_mode = fastAccumulationMode_ ? 0 : 1;
    CUBLAS_THROW_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc,
        hipblasLtMatmulDescAttributes_t::CUBLASLT_MATMUL_DESC_FAST_ACCUM,
        &ifast_accumulation_mode, sizeof(ifast_accumulation_mode)));
    p_scale_a = scale_A.GetTensorRawData();
    CUBLAS_THROW_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &p_scale_a,
        sizeof(p_scale_a)));
    p_scale_b = scale_B.GetTensorRawData();
    p_scale_before = p_scale_b;
    CUBLAS_THROW_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &p_scale_b,
        sizeof(p_scale_b)));

    // float 8
    if (out_dtype == ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT8E4M3FN ||
        out_dtype == ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT8E5M2) {
      std::vector<int64_t> scale_dimensions{1};
      Ort::UnownedValue scale_Y = ctx.GetOutput(1, scale_dimensions);
      p_scale_d = scale_Y.GetTensorRawData();
      CUBLAS_THROW_IF_ERROR(hipblasLtMatmulDescSetAttribute(
          operationDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &p_scale_d,
          sizeof(p_scale_d)));
      auto memsY = scale_Y.GetTensorMemoryInfo();
      EXT_ENFORCE(memsY.GetDeviceType() ==
                      OrtMemoryInfoDeviceType::OrtMemoryInfoDeviceType_GPU,
                  "output scale B is not on CUDA");
      // For FP8 output, cuBLAS requires C_type to be same as bias_type
      CUBLAS_THROW_IF_ERROR(
          hipblasLtMatrixLayoutCreate(&Cdesc, bias_cuda_type, M, N, ldd));
      CUBLAS_THROW_IF_ERROR(hipblasLtMatmulDescSetAttribute(
          operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_DATA_TYPE, &bias_cuda_type,
          sizeof(bias_cuda_type)));
    } else {
      CUBLAS_THROW_IF_ERROR(
          hipblasLtMatrixLayoutCreate(&Cdesc, d_cuda_type, M, N, ldd));
    }
  } else {
    CUBLAS_THROW_IF_ERROR(
        hipblasLtMatrixLayoutCreate(&Cdesc, d_cuda_type, M, N, ldd));
  }

  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;
  hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE,
                                 &epilogue, sizeof(epilogue));

  // See
  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulPreferenceAttributes_t#cublasltmatmulpreferenceattributes-t
  // The workspace should be allocated once from OpKernelContext assuming
  // only one cuda function is running at a time (which is not necessarily true
  // with H100).
  size_t workspaceSize = std::max(
      (size_t)1 << 20,
      (std::min((size_t)(1 << 24), (size_t)std::max(K * M, K * N) * 4) +
       16)); // suggested fixed value 24Mb
  workspaceSize -= workspaceSize % 16;
  hipblasLtMatmulPreference_t preference = nullptr;
  hipblasLtMatmulPreferenceCreate(&preference);
  hipblasLtMatmulPreferenceSetAttribute(preference,
                                       HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                       &workspaceSize, sizeof(workspaceSize));

  if (row_major_) {
    hipblasLtOrder_t matrixOrder = HIPBLASLT_ORDER_ROW;
    CUBLAS_THROW_IF_ERROR(
        hipblasLtMatrixLayoutSetAttribute(Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                         &matrixOrder, sizeof(matrixOrder)));
    CUBLAS_THROW_IF_ERROR(
        hipblasLtMatrixLayoutSetAttribute(Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                         &matrixOrder, sizeof(matrixOrder)));
    CUBLAS_THROW_IF_ERROR(
        hipblasLtMatrixLayoutSetAttribute(Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                         &matrixOrder, sizeof(matrixOrder)));
    CUBLAS_THROW_IF_ERROR(
        hipblasLtMatrixLayoutSetAttribute(Ddesc, HIPBLASLT_MATRIX_LAYOUT_ORDER,
                                         &matrixOrder, sizeof(matrixOrder)));
  }

  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulAlgoGetHeuristic#cublasltmatmulalgogetheuristic
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  int returnedResults = 0;
  hipblasStatus_t cuda_status = hipblasLtMatmulAlgoGetHeuristic(
      cublasLt, operationDesc, Adesc, Bdesc, Cdesc, Ddesc, preference, 1,
      &heuristicResult, &returnedResults);
  EXT_ENFORCE(returnedResults > 0 && cuda_status == HIPBLAS_STATUS_SUCCESS,
              " Unable to find any suitable algorithm due to ",
              cublasGetErrorEnum(cuda_status), ", preference=", preference,
              ", returnedResults=", returnedResults, ", alpha=", alpha_,
              // ", beta=", beta_,
              ", n_inputs=", n_inputs,
              ", A_type=", CudaDataTypeToString(a_cuda_type),
              ", B_type=", CudaDataTypeToString(b_cuda_type),
              ", result_type=", CudaDataTypeToString(d_cuda_type),
              ", bias_type=", CudaDataTypeToString(bias_cuda_type),
              ", scale_type=", CudaDataTypeToString(scale_cuda_type),
              ", computeType=", CublasComputeTypeToString(computeType_),
              ", epilogue=", epilogue, ", smCount=", smCount_,
              ", transA=", transA_, ", transB=", transB_,
              ", fastAccumulationMode=", (fastAccumulationMode_ ? 1 : 0),
              ", M=", M, ", N=", N, ", K=", K, ", lda=", lda, ", ldb=", ldb,
              ", ldd=", ldd, ", workspaceSize=", workspaceSize,
              ". Check NVIDIA documentation to see what combination is valid: ",
              "https://docs.nvidia.com/cuda/cublas/"
              "index.html?highlight=hipblasLtMatmulAlgoGetHeuristic#"
              "cublasltmatmulalgogetheuristic.");
  void *workspace = nullptr;
  if (workspaceSize > 0) {
    hipMalloc((void **)&workspace, workspaceSize);
  }
  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmul#cublasltmatmul
  float beta = 0;
  CUBLAS_THROW_IF_ERROR(hipblasLtMatmul(
      cublasLt, operationDesc, static_cast<const void *>(&alpha_), /* alpha */
      input_A.GetTensorRawData(),                                  /* A */
      Adesc, input_B.GetTensorRawData(),                           /* B */
      Bdesc, static_cast<const void *>(&beta),                     /* beta */
      nullptr,                                                     /* C */
      Cdesc, Y.GetTensorMutableRawData(),                          /* Y */
      Ddesc, &heuristicResult.algo,                                /* algo */
      workspace,               /* workspace */
      workspaceSize, stream)); /* stream */
  if (workspaceSize > 0) {
    hipFree(workspace);
  }

  hipblasLtMatmulPreferenceDestroy(preference);
  hipblasLtMatrixLayoutDestroy(Ddesc);
  hipblasLtMatrixLayoutDestroy(Cdesc);
  hipblasLtMatrixLayoutDestroy(Bdesc);
  hipblasLtMatrixLayoutDestroy(Adesc);
  hipblasLtMatmulDescDestroy(operationDesc);
  CUBLAS_THROW_IF_ERROR(hipblasLtDestroy(cublasLt));
  EXT_ENFORCE(p_scale_d == p_scale_before, "Output scale needs to be copied.");
}

} // namespace ortops
