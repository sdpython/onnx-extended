#include "hip/hip_runtime.h"
#include "common/common_kernels_cuda.h"
#include "custom_gemm.h"
#include <hipblaslt.h>
#include <hipblas.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>

// see
// https://gitlab.com/nvidia/headers/cuda-individual/cublas/-/blob/main/hipblaslt.h

namespace ortops {

////////////////////
// CustomGemmOpFloat
////////////////////

void *CustomGemmOpFloat::CreateKernel(const OrtApi &api,
                                      const OrtKernelInfo *info) const {
  return std::make_unique<CustomGemmKernel>(api, info).release();
};

const char *CustomGemmOpFloat::GetName() const { return "CustomGemmFloat"; };

const char *CustomGemmOpFloat::GetExecutionProviderType() const {
  return "CUDAExecutionProvider";
};

size_t CustomGemmOpFloat::GetInputTypeCount() const { return 2; };

ONNXTensorElementDataType CustomGemmOpFloat::GetInputType(size_t index) const {
  return ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT;
};

size_t CustomGemmOpFloat::GetOutputTypeCount() const { return 1; };

ONNXTensorElementDataType CustomGemmOpFloat::GetOutputType(size_t index) const {
  return ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT;
};

///////////////////////////
// CustomGemmOpFloat8E4M3FN
///////////////////////////

#if ORT_VERSION >= 1160 && CUDA_VERSION >= 11080

void *CustomGemmOpFloat8E4M3FN::CreateKernel(const OrtApi &api,
                                             const OrtKernelInfo *info) const {
  return std::make_unique<CustomGemmKernel>(api, info).release();
};

const char *CustomGemmOpFloat8E4M3FN::GetName() const {
  return "CustomGemmFloat8E4M3FN";
};

const char *CustomGemmOpFloat8E4M3FN::GetExecutionProviderType() const {
  return "CUDAExecutionProvider";
};

size_t CustomGemmOpFloat8E4M3FN::GetInputTypeCount() const { return 4; };

ONNXTensorElementDataType
CustomGemmOpFloat8E4M3FN::GetInputType(size_t index) const {
  switch (index) {
  case 0: // A
  case 1: // B
    return ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT8E4M3FN;
  case 2: // scale A
  case 3: // scale B
    return ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT;
  default:
    EXT_THROW("index=", index, " is out of boundary.");
  }
};

size_t CustomGemmOpFloat8E4M3FN::GetOutputTypeCount() const { return 2; };

ONNXTensorElementDataType
CustomGemmOpFloat8E4M3FN::GetOutputType(size_t index) const {
  // D, scale D
  return ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT16;
};

#endif

///////////////////
// CustomGemmKernel
///////////////////

CustomGemmKernel::CustomGemmKernel(const OrtApi &api,
                                   const OrtKernelInfo *info) {
  ThrowOnError(api, api.KernelInfoGetAttribute_float(info, "alpha", &alpha_));
  // ThrowOnError(api, api.KernelInfoGetAttribute_float(info, "beta", &beta_));
  transA_ =
      KernelInfoGetOptionalAttributeInt64AsBool(api, info, "transA", false);
  transB_ =
      KernelInfoGetOptionalAttributeInt64AsBool(api, info, "transB", false);
  fastAccumulationMode_ = KernelInfoGetOptionalAttributeInt64AsBool(
      api, info, "fastAccumulationMode", true);
  smCount_ = KernelInfoGetOptionalAttributeInt64(api, info, "smCount", 0);

  // A string attribute.
  std::string compute_type = KernelInfoGetOptionalAttributeString(
      api, info, "computeType", "HIPBLAS_COMPUTE_32F");
  if (compute_type == "HIPBLAS_COMPUTE_16F") {
    computeType_ = HIPBLAS_COMPUTE_16F;
  } else if (compute_type == "HIPBLAS_COMPUTE_32F") {
    computeType_ = HIPBLAS_COMPUTE_32F;
  } else if (compute_type == "HIPBLAS_COMPUTE_32F_FAST_16F") {
    computeType_ = HIPBLAS_COMPUTE_32F_FAST_16F;
  } else if (compute_type == "HIPBLAS_COMPUTE_32F_FAST_16BF") {
    computeType_ = HIPBLAS_COMPUTE_32F_FAST_16BF;
  } else if (compute_type == "HIPBLAS_COMPUTE_32F_FAST_TF32") {
    computeType_ = HIPBLAS_COMPUTE_32F_FAST_TF32;
  } else {
    EXT_THROW("Unexpected value for compute_type '", compute_type, "'.");
  }
}

void CustomGemmKernel::set(int M, int N, int K, int &lda, int &ldb,
                           int &ldd) const {
  if (transA_ && !transB_) { // TN
    lda = K;
    ldb = K;
    ldd = M;
  } else if (!transA_ && !transB_) { // NN
    lda = M;
    ldb = K;
    ldd = M;
  } else if (!transA_ && transB_) { // NT
    lda = M;
    ldb = N;
    ldd = M;
  } else { // TT
    EXT_THROW("transA_ == true && transB_ == true not allowed.");
  }
}

void CustomGemmKernel::Compute(OrtKernelContext *context) {
  Ort::KernelContext ctx(context);
  Ort::ConstValue input_A = ctx.GetInput(0);
  Ort::ConstValue input_B = ctx.GetInput(1);
  Ort::ConstValue scale_A, scale_B;
  int n_inputs = ctx.GetInputCount();
  if (n_inputs == 4) {
    scale_A = ctx.GetInput(2);
    scale_B = ctx.GetInput(3);
  } else if (n_inputs != 2) {
    EXT_THROW("Number of inputs must be 2 or 4.");
  }

  std::vector<int64_t> a_shape = input_A.GetTensorTypeAndShapeInfo().GetShape();
  std::vector<int64_t> b_shape = input_B.GetTensorTypeAndShapeInfo().GetShape();

  EXT_ENFORCE(a_shape.size() == 2);
  EXT_ENFORCE(b_shape.size() == 2);

  ONNXTensorElementDataType dtypes[4] = {
      input_A.GetTensorTypeAndShapeInfo().GetElementType(),
      input_B.GetTensorTypeAndShapeInfo().GetElementType(),
      n_inputs == 4 ? scale_A.GetTensorTypeAndShapeInfo().GetElementType()
                    : ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT,
      n_inputs == 4 ? scale_B.GetTensorTypeAndShapeInfo().GetElementType()
                    : ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT,
  };

  int M, N, K;
  if (transA_) {
    M = a_shape[1];
    K = a_shape[0];
  } else {
    M = a_shape[0];
    K = a_shape[1];
  }

  N = transB_ ? b_shape[0] : b_shape[1];
  EXT_ENFORCE(M >= 0 && K > 0 && N >= 0);

  std::vector<int64_t> dimensions{M, N};
  Ort::UnownedValue Y = ctx.GetOutput(0, dimensions);
  ONNXTensorElementDataType out_dtype =
      Y.GetTensorTypeAndShapeInfo().GetElementType();

  hipStream_t stream = (hipStream_t)ctx.GetGPUComputeStream();
  hipblasLtHandle_t cublasLt;
  CUBLAS_THROW_IF_ERROR(hipblasLtCreate(&cublasLt));

  // #if defined(CUDA_VERSION) && CUDA_VERSION >= 11080
  int lda, ldb, ldd;
  set(M, N, K, lda, ldb, ldd);

  // Gemm, note that CUDA assumes col-major, so Y(N,M) = alpha * op(B) x op(A) +
  // beta * C
  hipblasLtMatmulDesc_t operationDesc = nullptr;
  hipblasLtMatrixLayout_t Adesc = nullptr, Bdesc = nullptr, Cdesc = nullptr,
                         Ddesc = nullptr;
  hipblasLtMatmulPreference_t preference = nullptr;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};

  hipblasLtOrder_t matrixOrder = HIPBLASLT_ORDER_ROW;
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create matrix descriptors. Not setting any extra attributes.
  hipDataType a_type = ToCudaDataType(dtypes[0]);
  hipDataType b_type = ToCudaDataType(dtypes[1]);
  hipDataType d_type = ToCudaDataType(out_dtype);
  hipDataType bias_type =
      ToCudaDataType(ONNX_TENSOR_ELEMENT_DATA_TYPE_FLOAT);
  hipDataType scale_dtype = bias_type;

  CUBLAS_THROW_IF_ERROR(hipblasLtMatrixLayoutCreate(
      &Adesc, a_type, transA_ ? M : K, transA_ ? K : M, lda));
  CUBLAS_THROW_IF_ERROR(hipblasLtMatrixLayoutCreate(
      &Bdesc, b_type, transB_ ? K : N, transB_ ? N : K, ldb));
  CUBLAS_THROW_IF_ERROR(
      hipblasLtMatrixLayoutCreate(&Cdesc, bias_type, M, N, ldd));
  CUBLAS_THROW_IF_ERROR(hipblasLtMatrixLayoutCreate(&Ddesc, d_type, M, N, ldd));

  CUBLAS_THROW_IF_ERROR(hipblasLtMatrixLayoutSetAttribute(
      Adesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &matrixOrder, sizeof(matrixOrder)));
  CUBLAS_THROW_IF_ERROR(hipblasLtMatrixLayoutSetAttribute(
      Bdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &matrixOrder, sizeof(matrixOrder)));
  CUBLAS_THROW_IF_ERROR(hipblasLtMatrixLayoutSetAttribute(
      Cdesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &matrixOrder, sizeof(matrixOrder)));
  CUBLAS_THROW_IF_ERROR(hipblasLtMatrixLayoutSetAttribute(
      Ddesc, HIPBLASLT_MATRIX_LAYOUT_ORDER, &matrixOrder, sizeof(matrixOrder)));

  if (n_inputs == 4) {
    // gemm float 8

    CUBLAS_THROW_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER,
        scale_A.GetTensorRawData(), sizeof(float)));
    CUBLAS_THROW_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER,
        scale_B.GetTensorRawData(), sizeof(float)));
  }

  hipblasLtMatmulDescCreate(&operationDesc, computeType_, scale_dtype);
  hipblasOperation_t transa = transA_ ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t transb = transB_ ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA,
                                 &transa, sizeof(transa));
  hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB,
                                 &transb, sizeof(transb));
  const int8_t ifast_accumulation_mode = fastAccumulationMode_ ? 0 : 1;
  hipblasLtMatmulDescSetAttribute(
      operationDesc,
      hipblasLtMatmulDescAttributes_t::CUBLASLT_MATMUL_DESC_FAST_ACCUM,
      &ifast_accumulation_mode, sizeof(ifast_accumulation_mode));
  /*
  if (has_C) {
    hipblasLtMatmulDescSetAttribute(operationDesc,
                                   hipblasLtMatmulDescAttributes_t::HIPBLASLT_MATMUL_DESC_BIAS_DATA_TYPE,
                                   &bias_type,
                                   sizeof(bias_type));
  */

  if (n_inputs == 4) {
    // float 8
    std::vector<int64_t> scale_dimensions{1};
    Ort::UnownedValue scale_Y = ctx.GetOutput(1, scale_dimensions);
    CUBLAS_THROW_IF_ERROR(hipblasLtMatmulDescSetAttribute(
        operationDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER,
        scale_Y.GetTensorRawData(), sizeof(float)));
  }

  if (smCount_ != 0) {
    int math_sm_count = static_cast<int>(smCount_);
    hipblasLtMatmulDescSetAttribute(operationDesc,
                                   CUBLASLT_MATMUL_DESC_SM_COUNT_TARGET,
                                   &math_sm_count, sizeof(math_sm_count));
  }

  hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE,
                                 &epilogue, sizeof(epilogue));

  hipblasLtMatmulPreferenceCreate(&preference);

  // See
  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulPreferenceAttributes_t#cublasltmatmulpreferenceattributes-t
  // The workspace should be allocated once from OpKernelContext assuming
  // only one cuda function is running at a time (which is not necessarily true
  // with H100). size_t type_size = std::max(std::max(TypeSize(dtypes[0]),
  // TypeSize(dtypes[1])), std::max(std::max(TypeSize(dtypes[2]),
  // TypeSize(dtypes[3])), TypeSize(dtypes[4])));
  size_t workspaceSize = std::max(
      (size_t)1 << 20,
      (std::min((size_t)(1 << 24), (size_t)std::max(K * M, K * N) * 4) +
       16)); // suggested fixed value 24Mb
  workspaceSize -= workspaceSize % 16;
  hipblasLtMatmulPreferenceSetAttribute(preference,
                                       HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES,
                                       &workspaceSize, sizeof(workspaceSize));

  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmulAlgoGetHeuristic#cublasltmatmulalgogetheuristic
  int returnedResults = 0;
  hipblasStatus_t cuda_status = hipblasLtMatmulAlgoGetHeuristic(
      cublasLt, operationDesc, Adesc, Bdesc, Cdesc, Ddesc, preference, 1,
      &heuristicResult, &returnedResults);
  EXT_ENFORCE(returnedResults > 0 && cuda_status == HIPBLAS_STATUS_SUCCESS,
              " Unable to find any suitable algorithm due to ",
              cublasGetErrorEnum(cuda_status), ", preference=", preference,
              ", returnedResults=", returnedResults, ", alpha=", alpha_,
              // ", beta=", beta_,
              ", A_type=", CudaDataTypeToString(ToCudaDataType(dtypes[0])),
              ", B_type=", CudaDataTypeToString(ToCudaDataType(dtypes[1])),
              ", C_type=", CudaDataTypeToString(ToCudaDataType(dtypes[2])),
              ", result_type=", CudaDataTypeToString(ToCudaDataType(dtypes[4])),
              ", bias_type=", CudaDataTypeToString(bias_type),
              ", scale_type=", CudaDataTypeToString(scale_dtype),
              ", computeType=", CublasComputeTypeToString(computeType_),
              ", epilogue=", epilogue, ", smCount=", smCount_,
              ", transA=", transA_, ", transB=", transB_,
              ", fastAccumulationMode=", (fastAccumulationMode_ ? 1 : 0),
              ", M=", M, ", N=", N, ", K=", K, ", lda=", lda, ", ldb=", ldb,
              ", ldd=", ldd, ", workspaceSize=", workspaceSize,
              ". Check NVIDIA documentation to see what combination is valid: ",
              "https://docs.nvidia.com/cuda/cublas/"
              "index.html?highlight=hipblasLtMatmulAlgoGetHeuristic#"
              "cublasltmatmulalgogetheuristic.");
  void *workspace = nullptr;
  if (workspaceSize > 0) {
    hipMalloc((void **)&workspace, workspaceSize);
  }
  // https://docs.nvidia.com/cuda/cublas/index.html?highlight=hipblasLtMatmul#cublasltmatmul
  float beta = 0;
  CUBLAS_THROW_IF_ERROR(hipblasLtMatmul(
      cublasLt, operationDesc, static_cast<const void *>(&alpha_),  /* alpha */
      input_A.GetTensorRawData(),                                   /* A */
      Adesc, input_B.GetTensorRawData(),                            /* B */
      Bdesc, static_cast<const void *>(&beta),                      /* beta */
      nullptr,                                                      /* C */
      Cdesc, Y.GetTensorMutableRawData(),                           /* Y */
      Ddesc, &heuristicResult.algo,                                 /* algo */
      workspace,                                                    /* workspace */
      workspaceSize, stream));                                      /* stream */
  if (workspaceSize > 0) {
    hipFree(workspace);
  }

  hipblasLtMatmulPreferenceDestroy(preference);
  hipblasLtMatrixLayoutDestroy(Ddesc);
  hipblasLtMatrixLayoutDestroy(Cdesc);
  hipblasLtMatrixLayoutDestroy(Bdesc);
  hipblasLtMatrixLayoutDestroy(Adesc);
  hipblasLtMatmulDescDestroy(operationDesc);
  CUBLAS_THROW_IF_ERROR(hipblasLtDestroy(cublasLt));
}

} // namespace ortops
